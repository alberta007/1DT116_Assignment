#include "hip/hip_runtime.h"
// tick_cuda.cu
#ifndef NOCUDA
#include <hip/hip_runtime.h>
#include <math.h>
#include "tick_cuda.h"  // Declaration header (if needed)
#include <cstdio>
#endif

// CUDA Kernel: Each thread handles one agent.
__global__ void tickCuda_kernel(
    int *agent_x, int *agent_y,
    float *destX, float *destY, float *destR,
    int *currentWaypointIndex,
    const float *waypoint_x, const float *waypoint_y, const float *waypoint_r,
    int numAgents, int numWaypoints)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numAgents) return;

    // Load current agent position.
    float ax = (float)agent_x[idx];
    float ay = (float)agent_y[idx];

    // Load current destination from the agent's SoA.
    float wx = destX[idx];
    float wy = destY[idx];
    float wr = destR[idx];

    // Load the current waypoint index.
    int wpIdx = currentWaypointIndex[idx];

    // Compute squared distance to destination.
    float dx = wx - ax;
    float dy = wy - ay;
    float dist2 = dx * dx + dy * dy;
    float wr2 = wr * wr;

    // If the agent is within the destination radius, update its waypoint.
    if (dist2 < wr2) {
        wpIdx = (wpIdx + 1) % numWaypoints;
        currentWaypointIndex[idx] = wpIdx;
        // Update destination values from global waypoint data.
        wx = waypoint_x[wpIdx];
        wy = waypoint_y[wpIdx];
        wr = waypoint_r[wpIdx];
        destX[idx] = wx;
        destY[idx] = wy;
        destR[idx] = wr;
        // Recompute differences for the movement step.
        dx = wx - ax;
        dy = wy - ay;
        dist2 = dx * dx + dy * dy;
    }

    // Compute Euclidean distance.
    float len = sqrtf(dist2);
    if (len < 1e-6f) len = 1.0f;  // Avoid division by zero.

    // Compute unit vector toward the destination.
    float nx = dx / len;
    float ny = dy / len;

    // Update position by one unit step.
    ax += nx;
    ay += ny;
    agent_x[idx] = (int)roundf(ax);
    agent_y[idx] = (int)roundf(ay); 
}

// Host function: Launches the CUDA kernel.
void tickCuda(AgentsSoA &agents, const WaypointsSoA &waypoints) {
#ifndef NOCUDA
    float *d_agents_destX, *d_agents_destY, *d_agents_destR;
    int *d_agents_currentWaypointIndex, *d_agents_x, *d_agents_y;
    float *d_waypoints_x, *d_waypoints_y, *d_waypoints_r;
    
    int numAgents = agents.x.size();
    int numWaypoints = waypoints.x.size();
    
    hipMalloc((void**)&d_agents_x, numAgents * sizeof(int));
    hipMalloc((void**)&d_agents_y, numAgents * sizeof(int));
    hipMalloc((void**)&d_agents_destX, numAgents * sizeof(float));
    hipMalloc((void**)&d_agents_destY, numAgents * sizeof(float));
    hipMalloc((void**)&d_agents_destR, numAgents * sizeof(float));
    hipMalloc((void**)&d_agents_currentWaypointIndex, numAgents * sizeof(int));

    hipMalloc((void**)&d_waypoints_x, numWaypoints * sizeof(float));
    hipMalloc((void**)&d_waypoints_y, numWaypoints * sizeof(float));
    hipMalloc((void**)&d_waypoints_r, numWaypoints * sizeof(float));

    hipMemcpy(d_agents_x, agents.x.data(), numAgents * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_agents_y, agents.y.data(), numAgents * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_agents_destX, agents.destX.data(), numAgents * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_agents_destY, agents.destY.data(), numAgents * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_agents_destR, agents.destR.data(), numAgents * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_agents_currentWaypointIndex, agents.currentWaypointIndex.data(), numAgents * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_waypoints_x, waypoints.x.data(), numWaypoints * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_waypoints_y, waypoints.y.data(), numWaypoints * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_waypoints_r, waypoints.r.data(), numWaypoints * sizeof(float), hipMemcpyHostToDevice);

    // Choose a block size and calculate grid size.
    int blockSize = 256;
    int gridSize = (numAgents + blockSize - 1) / blockSize;

    // Launch the kernel.
    tickCuda_kernel<<<gridSize, blockSize>>>(
        d_agents_x, d_agents_y,
        d_agents_destX, d_agents_destY, d_agents_destR,
        d_agents_currentWaypointIndex,
        d_waypoints_x, d_waypoints_y, d_waypoints_r,
        numAgents, numWaypoints
    );

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Check for execution errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error after synchronization: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(agents.x.data(), d_agents_x, numAgents * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(agents.y.data(), d_agents_y, numAgents * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(agents.destX.data(), d_agents_destX, numAgents * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(agents.destY.data(), d_agents_destY, numAgents * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(agents.destR.data(), d_agents_destR, numAgents * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(agents.currentWaypointIndex.data(), d_agents_currentWaypointIndex, numAgents * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy((float *)waypoints.x.data(), d_waypoints_x, numWaypoints * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((float *)waypoints.y.data(), d_waypoints_y, numWaypoints * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy((float *)waypoints.r.data(), d_waypoints_r, numWaypoints * sizeof(float), hipMemcpyDeviceToHost);

    // Synchronize the device to ensure the kernel has finished.
    hipFree(d_agents_x);
    hipFree(d_agents_y);
    hipFree(d_agents_destX);
    hipFree(d_agents_destY);
    hipFree(d_agents_destR);
    hipFree(d_agents_currentWaypointIndex);
    hipFree(d_waypoints_x);
    hipFree(d_waypoints_y);
    hipFree(d_waypoints_r);
#endif
}
