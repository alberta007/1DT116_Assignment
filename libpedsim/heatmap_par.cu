#include "hip/hip_runtime.h"
#ifndef NOCUDA

#include "heatmap_par.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <cstdio>

#endif

#define SIZE 1024
#define CELLSIZE 5
#define SCALED_SIZE (SIZE * CELLSIZE)
#define WEIGHTSUM 273


__global__ void updateHeatmapPar(int *heatmap, int *scaled_heatmap, int *blurred_heatmap, int sizeOfAgents, int *d_agents_desX, int *d_agents_desY)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= SIZE || y >= SIZE) return;

    if (x < SIZE && y < SIZE) {
        int idx = y * SIZE + x;
        heatmap[idx] = (int)roundf(heatmap[idx] * 0.8);
    }
	
    if (x < sizeOfAgents) {
        int x1 = d_agents_desX[x];
        int y1 = d_agents_desY[x];
        if (x1 >= 0 && x1 < SIZE && y1 >= 0 && y1 < SIZE) {
            int idx = y1 * SIZE + x1;
            atomicAdd(&heatmap[idx], 40);
        }
    }


    int idx = y * SIZE + x;
    if(heatmap[idx] > 255) {
        heatmap[idx] = 255;
    }

    int scaledSIZE = SIZE * CELLSIZE;
    if (x < scaledSIZE && y < scaledSIZE) {
        int origX = x / CELLSIZE;
        int origY = y / CELLSIZE;
        scaled_heatmap[y * scaledSIZE + x] = heatmap[origY * SIZE + origX];
    }
	
    extern __shared__ int s_data[];
    int s_SIZE = blockDim.x + 4; 
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int gx = blockIdx.x * blockDim.x + tx;
    int gy = blockIdx.y * blockDim.y + ty;
    
    for (int j = ty; j < blockDim.y + 4; j += blockDim.y) {
        for (int i = tx; i < blockDim.x + 4; i += blockDim.x) {
            int global_x = blockIdx.x * blockDim.x + i - 2; // subtract halo offset
            int global_y = blockIdx.y * blockDim.y + j - 2;
            if (global_x >= 0 && global_x < SCALED_SIZE && global_y >= 0 && global_y < SCALED_SIZE)
                s_data[j * s_SIZE + i] = scaled_heatmap[global_y * SCALED_SIZE + global_x];
            else
                s_data[j * s_SIZE + i] = 0;
        }
    }
    __syncthreads();
    if (gx >= 2 && gx < SCALED_SIZE - 2 && gy >= 2 && gy < SCALED_SIZE - 2) {
        int sum = 0;
        int w[5][5] = {
            { 1,  4,  7,  4, 1 },
            { 4, 16, 26, 16, 4 },
            { 7, 26, 41, 26, 7 },
            { 4, 16, 26, 16, 4 },
            { 1,  4,  7,  4, 1 }
        };
        for (int ky = 0; ky < 5; ky++) {
            for (int kx = 0; kx < 5; kx++) {
                int s_val = s_data[(ty + ky) * s_SIZE + (tx + kx)];
                sum += w[ky][kx] * s_val;
            }
        }
        int value = sum / WEIGHTSUM;

        blurred_heatmap[gy * SCALED_SIZE + gx] = 0x00FF0000 | (value << 24);        
    }
}

// Updates the heatmap according to the agent positions
__host__ void updateHeatMapCuda(int *heatmap, int *scaled_heatmap, int *blurred_heatmap, int *desiredXs, int *desiredYs, int numAgents) {
#ifndef NOCUDA
    int *d_heatmap, *d_scaled_heatmap, *d_blurred_heatmap;
    int *d_agents_desX, *d_agents_desY;

    size_t heatmapBytes = SIZE * SIZE * sizeof(int);
    size_t scaledBytes  = SCALED_SIZE * SCALED_SIZE * sizeof(int);
    size_t agentsBytes  = numAgents * sizeof(int);
   

    hipMalloc((void**)&d_heatmap, heatmapBytes);
    hipMalloc((void**)&d_scaled_heatmap, scaledBytes);
    hipMalloc((void**)&d_blurred_heatmap, scaledBytes);
    hipMalloc((void**)&d_agents_desX, agentsBytes);
    hipMalloc((void**)&d_agents_desY, agentsBytes);

    hipMemcpy(d_blurred_heatmap, blurred_heatmap, scaledBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_scaled_heatmap, scaled_heatmap, scaledBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_heatmap, heatmap, heatmapBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_agents_desX, desiredXs, agentsBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_agents_desY, desiredYs, agentsBytes, hipMemcpyHostToDevice);


    dim3 blockDim(16, 16);
    dim3 gridDim((SIZE + blockDim.x - 1) / blockDim.x, (SIZE + blockDim.y - 1) / blockDim.y);
    size_t sharedMemSize = (blockDim.x + 4) * (blockDim.y + 4) * sizeof(int);

    updateHeatmapPar<<<gridDim, blockDim, sharedMemSize>>>(d_heatmap, d_scaled_heatmap, d_blurred_heatmap, numAgents, d_agents_desX, d_agents_desY);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();


    hipMemcpy(heatmap, d_heatmap, heatmapBytes, hipMemcpyDeviceToHost);
    hipMemcpy(scaled_heatmap, d_scaled_heatmap, scaledBytes, hipMemcpyDeviceToHost);
    hipMemcpy(blurred_heatmap, d_blurred_heatmap, scaledBytes, hipMemcpyDeviceToHost);

    hipFree(d_heatmap);
    hipFree(d_scaled_heatmap);
    hipFree(d_blurred_heatmap);
    hipFree(d_agents_desX);
    hipFree(d_agents_desY);

#endif
}